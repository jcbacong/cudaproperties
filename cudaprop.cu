
#include <hip/hip_runtime.h>
#include <stdio.h>

int getSPcores(hipDeviceProp_t devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if (devProp.minor == 1) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta
      if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}


int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Number of CUDA cores: %d\n", getSPcores(prop));
    printf("  Warp Size: %d\n", prop.warpSize);
    printf("  Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  Max Grid Size: (%d, %d, %d)\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    printf("  Max Block Size: (%d, %d, %d)\n", prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
    printf("  Max Threads Per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Total Global Memory: %zu GB\n", prop.totalGlobalMem>>30);
    printf("  Shared Memory per Block: %zu kB\n", prop.sharedMemPerBlock>>10);
    printf("  Register per Block: %d kB \n\n", prop.regsPerBlock/1024);

  }
}